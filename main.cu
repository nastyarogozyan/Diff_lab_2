#include<hip/hip_runtime.h>
#include <cmath>
#include <cstdlib>
#include <cstdio>
#include <cinttypes>
#include <fstream>
#define PI 3.14159265358979323846
 
const double delta = 0.005;
const double delta_r = 0.01;
const double eps = 1e-13;
 
const uint8_t m = 100; 
const uint16_t p = 2000;
const double g = 9.8;
 
double Ax = -0.353, Bx = 0.353, Ay = 0.3, By = Ay, C = 3 * PI / 8;
 
struct Diff_questions {
  double Ax;
  double Bx;
  double Ay;
  double By;
  double C;
  double Pi;
  double Delta;
  double Eps;
} typedef Diff ;
 
__device__ double Cal_dist(double* x0, double* x1, size_t n) {
  double sum = 0;
  for (size_t i = 0; i < n; ++i) {
    sum += std::pow(x0[i] - x1[i], 2);
  }
  return std::sqrt(sum);
}
 
__device__ void Cal_F(double* result, double* input, Diff* diff) {
   __shared__ double Arr[5];
  if (threadIdx.x == 0) {
    Arr[0] = input[0] + input[2] * std::cos(3 * diff->Pi / 2 - input[3]) - diff->Ax;
  } else if (threadIdx.x == 1) {
    Arr[1] = input[1] + input[2] * std::cos(3 * diff->Pi / 2 + input[4]) - diff->Bx;
  } else if (threadIdx.x == 2) {
    Arr[2] = input[2] + input[2] * std::sin(3 * diff->Pi / 2 - input[3]) - diff->Ay;
  } else if (threadIdx.x == 3) {
    Arr[3] = (input[3] + input[4]) * input[2] + (input[1] - input[0]) - diff->C;
  } else if (threadIdx.x == 4) {
    Arr[4] = input[2] + input[2] * std::sin(3 * diff->Pi / 2 + input[4]) - diff->By;
  }
 
  __syncthreads();
  if (threadIdx.x == 0) {
    memcpy(result, Arr, sizeof(Arr));
  }
  __syncthreads();
}
 
__global__ void Cal_val(double* x0, double* x1, Diff* diff, size_t n) {
  __shared__ unsigned count;
  double* NewV = new double[5];
  while(true) {
    Cal_F(NewV, x0, diff);
    __syncthreads();
    if (threadIdx.x == 0) {
      for (size_t i = 0; i < n; ++i) {
        x1[i] = x0[i] - NewV[i] * diff->Delta;
      }
    }
    __syncthreads();
    if (threadIdx.x == 0) atomicAdd(&count, 1);
    __syncthreads();
    if (Cal_dist(x0, x1, n) < diff->Eps) break;
    if (threadIdx.x == 0) {
      for (size_t i = 0; i < n; ++i) {
        x0[i] = x1[i];
      }
    }
    __syncthreads();
  }
  delete[] NewV ;
}
 
__host__ void print_result(double* x) {
    printf("x1 : %lf\n", x[0]);
    printf("x2 : %lf\n", x[1]);
    printf("y : %lf\n", x[2]);
    printf("phi1 : %lf\n", x[3]);
    printf("phi2 : %lf\n", x[4]);
    printf("F(x) = {%.10e, %.10e, %.10e, %.10e, %.10e}\n",
      x[0] + x[2] * std::cos(1.5 * PI - x[3]) - Ax,
      x[1] + x[2] * std::cos(1.5 * PI + x[4]) - Bx,
      x[2] + x[2] * std::sin(1.5 * PI - x[3]) - Ay,
      (x[3] + x[4]) * x[2] + (x[1] - x[0]) - C,
      x[2] + x[2] * std::sin(1.5 * PI + x[4]) - By
    );
}
 
int main() {
  Diff* diff;
  const int Questions = 5;
  double *x0, *x1;
  hipMallocManaged(&x0, sizeof(double) * Questions );
  hipMallocManaged(&x1, sizeof(double) * Questions );
  x0[0] = -0.1; x0[1] = 0.1; x0[2] = 0.0; x0[3] = 2.0; x0[4] = 2.0;
  x1[0] = 0.0; x1[1] = 0.0; x1[2] = 0.0; x1[3] = 0.0; x1[4] = 0.0;
  hipMallocManaged(&diff, sizeof(Diff));
  diff->Ax = Ax, diff->Ay = Ay, 
  diff->Bx = Bx, diff->By = By, 
  diff->C = C, diff->Pi = PI, 
  diff->Delta = delta, diff->Eps = eps;
 
  double Vs = 0;
  int Blocks_ = 1;
  int Threads_Blocks = 5;
  for (double t = 0; t <= 2.5; t += delta_r) {
    Cal_val<<<Blocks_ , Threads_Blocks >>>(x0, x1, diff, Questions);
    hipDeviceSynchronize();
    print_result(x0);
    Vs += (p * (x1[1] - x1[0]) - m * g) * delta_r/ m;
    diff->Ay += Vs * delta_r;
    diff->By = diff->Ay;
  }
  hipFree(&x0);
  hipFree(&x1);
  return 0;
}